#include <iostream>
#include <cstddef>
#include <hip/hip_runtime.h>


void printCurrentDeviceProperties() {
    int currentDevice;
    hipGetDevice(&currentDevice);  // Get the current device ID

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, currentDevice);  // Get properties of the current device

    std::cout << "Currently Active CUDA Device: " << currentDevice << "\n";
    std::cout << "Device Name: " << prop.name << "\n";
    std::cout << "CUDA Capability: " << prop.major << "." << prop.minor << "\n";
    std::cout << "Total Global Memory: " << prop.totalGlobalMem / (1024.0 * 1024.0) << " MB\n";
    std::cout << "Shared Memory per Block: " << prop.sharedMemPerBlock / 1024.0 << " KB\n";
    std::cout << "Warp Size: " << prop.warpSize << "\n";
    std::cout << "Max Threads per Block: " << prop.maxThreadsPerBlock << "\n";
}


// online version of softmax, which uses parallleism across rows to compute softmax
__global__ void soft_max_kernel_shared_memory(const float* matrix, float* matrix_out, size_t columns, size_t N) {
    
    extern __shared__ float sdata[]; // this will be blockdim * 2; 
    size_t row_index = blockIdx.x;
    size_t id = threadIdx.x;
    size_t range = (columns + blockDim.x-1)/blockDim.x;
    size_t block_dim = blockDim.x;

    if(id < block_dim){ 
        float max_num = matrix[row_index * columns + id];
        float sum  = 1.0f; 
        for(int i = 1; i < range; i++){
            int index = row_index * columns  + i*block_dim + id;
            if(index < (row_index + 1) * columns){
                float new_max_num = fmaxf(max_num, matrix[index]);
                sum = sum * expf(max_num - new_max_num) + expf(matrix[index] - new_max_num);
                max_num = new_max_num;
            }
        }
        sdata[id] = max_num; 
        sdata[id + block_dim] = sum;
    }

    __syncthreads();
    // we are done with calculating local max and local sums for the threads  
    // now reduce them
    int length = min(block_dim, columns); // the number of threads that are active to reduce

    while (length > 1){
        int old_length = length;
        length = (length+1)/2;
        if(id < length  && (old_length-id-1 > id)){
            // id and old_length -id -1
            float new_max = fmaxf(sdata[id], sdata[old_length-id-1]);
            float new_sum = sdata[id+ block_dim] * expf(sdata[id] - new_max) + sdata[old_length-id-1 + block_dim] * expf(sdata[old_length-id-1] - new_max);
            sdata[id] = new_max;
            sdata[id + block_dim] = new_sum;
        }
        __syncthreads();
    }

    if(id < block_dim){ 
        for(int i = 0; i < range; i++){
            int index = row_index * columns  + i*block_dim + id;
            if(index < (row_index + 1) * columns){
                matrix_out[index] = expf(matrix[index] - sdata[0])/sdata[block_dim];
            }
        }
    }
}

int main() {
    printCurrentDeviceProperties();
    size_t rows = 1024;
    size_t columns = 1024 * 64;
    size_t size = rows * columns * sizeof(float);
    // ideally this should be configured on the basis of the column size of the matrix, but the code is written agnostic to block_size
    int block_size = 256; 
    int count = rows * columns;
    float* matrix_cpu = (float*) malloc(size) ;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Seed the random number generator
    srand(time(NULL));

    // Fill the array with random numbers from -5 to 5
    for (size_t i = 0; i < rows * columns; ++i) {
        matrix_cpu[i] = -5 + ((float)rand() / RAND_MAX) * 10;
    }

    float* matrix;
    hipMalloc(&matrix, rows * columns * sizeof(float));
    hipMemcpy(matrix, matrix_cpu, size, hipMemcpyHostToDevice);
    
    // the output matrix
    float* matrix_out;
    hipMalloc(&matrix_out, rows * columns * sizeof(float));
    
    hipEventRecord(start);
    soft_max_kernel_shared_memory<<<rows, block_size, block_size * 2 * sizeof(float)>>>(matrix, matrix_out, columns, count);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float elapsed = 0.0f;
    hipEventElapsedTime(&elapsed, start, stop);
    std::cout << "Softmax kernel took " << elapsed << " ms\n";

    float* matrix_cpu_out = (float*) malloc(1024*64*4) ;
    hipMemcpy(matrix_cpu_out, matrix_out, 1024*64*4, hipMemcpyDeviceToHost);
    float sum = 0.0f;
    for (int i = 0; i < 1024*64; ++i)
        sum += matrix_cpu_out[i];
    std::cout << "The sum of the first column is: '" << sum << std::endl;
    hipFree(matrix);
    hipFree(matrix_out);
    free(matrix_cpu_out);
    free(matrix_cpu);
    return 0;
}
